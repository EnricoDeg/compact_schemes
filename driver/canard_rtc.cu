#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

#include "common/parameters.hpp"
#include "common/data_types.hpp"
#include "cuda/check.hpp"
#include "cuda/driver.hpp"
#include "mpi/check.hpp"
#include "mpi/driver.hpp"
#include "numerics_rtc.hpp"

// Main program
int main()
{
    mpi_driver mpi_driver_instance{};
    cuda_driver cuda_driver_instance{};

    // Get the number of processes
    int world_size;
    check_mpi(MPI_Comm_size(MPI_COMM_WORLD, &world_size));

    // Get the rank of the process
    int world_rank;
    check_mpi(MPI_Comm_rank(MPI_COMM_WORLD, &world_rank));

    static constexpr unsigned int ax = 2;

    // Subdomain info
    t_dcomp dcomp_info;
    dcomp_info.lxi = 1024;
    dcomp_info.let = 64;
    dcomp_info.lze = 64;
    dcomp_info.lmx = dcomp_info.lxi * dcomp_info.let * dcomp_info.lze;

    float *infield = (float *)malloc(dcomp_info.lmx * sizeof(float));
    float *d_infield;
    check_cuda( hipMalloc(&d_infield, dcomp_info.lmx * sizeof(float)) );

    float *outfield = (float *)malloc(dcomp_info.lmx * sizeof(float));
    float *d_outfield;
    check_cuda( hipMalloc(&d_outfield, dcomp_info.lmx * sizeof(float)) );

    for(unsigned int i = 0; i < dcomp_info.lmx; ++i)
    {
        infield[i] = i;
    }
    check_cuda( hipMemcpy(d_infield, infield,
        dcomp_info.lmx * sizeof(float), hipMemcpyHostToDevice));

    int nstart = 0;
    int nend   = 0;

    auto numerics_instance = numerics_rtc<float>(dcomp_info);

    numerics_instance.template deriv1d_compile<ax>(dcomp_info, nstart, nend);

    numerics_instance.template fill_buffer_compile<ax>();

    constexpr unsigned int NStreams = 1;
    hipStream_t streams[NStreams];
    for(unsigned int i = 0; i < NStreams; ++i)
    {
       check_cuda_driver(hipStreamCreateWithFlags ( &streams[i], hipStreamNonBlocking ));
    }

    // setup derivatives
    numerics_instance.deriv_setup();

    check_mpi(MPI_Barrier(MPI_COMM_WORLD));

    for(unsigned int i = 0; i < 10; ++i)
    {
        numerics_instance.template deriv1d<ax>(d_infield,
            d_outfield,
            dcomp_info,
            0,
            streams);
    }

    for(unsigned int i = 0; i < NStreams; ++i)
    {
        check_cuda_driver( hipStreamDestroy ( streams[i] ));
    }

    check_cuda( hipMemcpy(outfield, d_outfield,
        dcomp_info.lmx * sizeof(float), hipMemcpyDeviceToHost) );

    float solution;
    if constexpr(ax == 0)
    {
        solution = infield[1] - infield[0];
    }
    else if constexpr(ax == 1)
    {
        solution = infield[dcomp_info.lxi] - infield[0];
    }
    else if constexpr(ax == 2)
    {
        solution = infield[dcomp_info.lxi * dcomp_info.let] - infield[0];
    }
    for(unsigned int i = 0; i < dcomp_info.lmx; ++i)
    {
        if(std::abs(outfield[i] - solution) / solution > 1e-6)
        {
            std::cout << std::abs(outfield[i] - solution) / solution << std::endl;
            std::cout << i << ": out = " << outfield[i] << " -- ref = " << solution << std::endl;
            exit(1);
        }
    }

    return 0;
}
