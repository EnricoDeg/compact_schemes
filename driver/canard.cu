#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

#include "common/parameters.hpp"
#include "numerics.hpp"
#include "gcbc.hpp"
#include "grid.hpp"
#include "physics.hpp"


// Main program
int main()
{
    // Initialize the MPI environment
    check_mpi(MPI_Init(NULL, NULL));

    // Get the number of processes
    int world_size;
    check_mpi(MPI_Comm_size(MPI_COMM_WORLD, &world_size));

    // Get the rank of the process
    int world_rank;
    check_mpi(MPI_Comm_rank(MPI_COMM_WORLD, &world_rank));

    static constexpr unsigned int ax = 0;

    // Subdomain info
    t_dcomp dcomp_info;
    dcomp_info.lxi = 128;
    dcomp_info.let = 256;
    dcomp_info.lze = 64;
    dcomp_info.lmx = dcomp_info.lxi * dcomp_info.let * dcomp_info.lze;

    // Spatial distance
    float h = 1.0;
    float h_1 = 1.0 / h;

    // cm
    float *d_cm0, *d_cm1, *d_cm2;
    hipMalloc(&d_cm0, 2 * NumberOfSpatialDims * dcomp_info.let * dcomp_info.lze * sizeof(float));
    hipMalloc(&d_cm1, 2 * NumberOfSpatialDims * dcomp_info.lxi * dcomp_info.lze * sizeof(float));
    hipMalloc(&d_cm2, 2 * NumberOfSpatialDims * dcomp_info.lxi * dcomp_info.let * sizeof(float));

    float *d_cm[3];
    d_cm[0] = d_cm0;
    d_cm[1] = d_cm1;
    d_cm[2] = d_cm2;

    // qa
    float * d_qa;
    hipMalloc(&d_qa, NumberOfSpatialDims * dcomp_info.lmx * sizeof(float));

    // de
    float * d_de;
    hipMalloc(&d_de, NumberOfSpatialDims * dcomp_info.lmx * sizeof(float));

    // pressure
    float * d_pressure;
    hipMalloc(&d_pressure, dcomp_info.lmx * sizeof(float));

    // yaco
    float *d_yaco;
    hipMalloc(&d_yaco, dcomp_info.lmx * sizeof(float));

    // ss
    float *d_ss;
    hipMalloc(&d_ss, dcomp_info.lmx * sizeof(float));

    // umf
    t_point<float> umf = {.x = 0.3, .y = 0.0, .z = 0.0 };

    // dudtmf
    t_point<float> dudtmf = {.x = 0.0, .y = 0.0, .z = 0.0 };

    // npex
    int * d_npex;
    hipMalloc(&d_npex, dcomp_info.lmx * sizeof(int));

    // ndf is 1 if halo exchange is needed, otherwise is 0
    unsigned int ndf[2][3];
    for(unsigned int ip = 0; ip < 2; ++ip)
    {
        for(unsigned int nn = 0; nn < 3; ++nn)
        {
            ndf[ip][nn] = 0;
        }
    }
    if(world_rank == 0)
    {
        ndf[1][ax] = 1;
    }
    else if(world_rank == 1)
    {
        ndf[0][ax] = 1;
    }

    // mcd indicates the pair process for each face. If there is no
    // halo exchange on a face, it is set to -1
    int mcd[2][3];
    for(unsigned int ip = 0; ip < 2; ++ip)
    {
        for(unsigned int nn = 0; nn < 3; ++nn)
        {
            mcd[ip][nn] = -1;
        }
    }

    if(world_rank == 0)
    {
        mcd[1][ax] = 1;
    }
    else if(world_rank == 1)
    {
        mcd[0][ax] = 0;
    }

    // nbc indicates the BC type for each face
    int nbc[2][3];
    for(unsigned int ip = 0; ip < 2; ++ip)
    {
        for(unsigned int nn = 0; nn < 3; ++nn)
        {
            nbc[ip][nn] = BC_PERIODIC;
        }
    }
    if(world_rank == 0)
    {
        nbc[0][ax] = BC_NON_REFLECTIVE;
        nbc[1][ax] = BC_INTER_SUBDOMAINS;
    }
    else if(world_rank == 1)
    {
        nbc[0][ax] = BC_INTER_SUBDOMAINS;
        nbc[1][ax] = BC_NON_REFLECTIVE;
    }

    auto grid_instance = grid<float>(dcomp_info);
    auto physics_instance = physics<true, float>(dcomp_info);

    auto numerics_instance = numerics<float>(dcomp_info);

    // setup derivatives
    numerics_instance.deriv_setup();

    hipStream_t stream[5];
    for(int i=0; i<5; i++) hipStreamCreate(&stream[i]);

    size_t n = 0;
    size_t ndt = 0;
    float dt = 0.1f;
    float dts = 0.0f;
    float dte = 0.0f;
    float timo = 0.0f;
    float dtsum = 0.0f;
    float tmax = 1.0;
    float cfl = 0.95f;
    int nout;
    float res;
    int ndati = -1;
    float dtk, dtko;
    int ndata = 2;
    bool output_enabled = false;

    physics_instance.init();

    check_mpi(MPI_Barrier(MPI_COMM_WORLD));

    do{
        std::cout << "Time step = " << n << std::endl;
        for(int nk = 0; nk < nkrk; ++nk)
        {

            // move frame velocity and acceleration before time advancing
            dtko = dt * min( max( nk - 2, 0 ), 1 ) / ( nkrk - nk + 3 );
            dtk  = dt * min( nk - 1, 1 ) / ( nkrk - nk + 2 );
            physics_instance.movef(dtko, dtk, timo);

            // temporary storage of primitive variables and pressure


            // compute time step size and output time
            if(nk == 1)
            {
                if(n % 10 == 1)
                {
                    ndt = n;
                    dts = dte;
                    physics_instance.calc_time_step(grid_instance.xim,
                                                    grid_instance.etm,
                                                    grid_instance.zem,
                                                    d_de,
                                                    d_yaco,
                                                    d_ss,
                                                    umf,
                                                    cfl,
                                                    &dte,
                                                    dcomp_info.lmx);
                }
                // dt = dts + (dte - dts) *
                //     std::sin(0.05f * pi * (n - ndt)) *
                //     std::sin(0.05f * pi * (n - ndt));

                nout = 0;
                res = (ndati + 1) * tmax / ndata;
                if((timo - res) * (timo + dt - res) <= 0.0f)
                {
                    nout = 1;
                    ndati++;
                }
            }

            // compute viscous shear stress
            physics_instance.calc_viscous_shear_stress(d_de, d_ss,
               grid_instance.xim, grid_instance.etm, grid_instance.zem,
               d_yaco, dcomp_info, h_1, ndf, mcd, &numerics_instance, &stream[0]);

            // compute fluxes
            physics_instance.calc_fluxes(d_qa, d_pressure, d_de,
                                         grid_instance.xim, grid_instance.etm, grid_instance.zem,
                                         dcomp_info, umf,
                                         h_1, ndf, mcd, &numerics_instance, &stream[0]);

            float dtwi = 1 / dt;

            // GCBC
            // auto gcbc_instance = gcbc<float, int>(dcomp_info);
            // gcbc_go(numerics_instance.drva_buffer, d_cm, gcbc_instance.drvb,
            //         d_qa, d_de, d_pressure, d_yaco, gcbc_instance.sbcc,
            //         umf, dudtmf, dcomp_info, dtwi,
            //         nbc, mcd);

            // sponge condition

            // update conservative variables
            dtko = dt * min(nk-1, 1) / (nkrk - nk + 2);
            dtk  = dt / (nkrk - nk + 1);
            physics_instance.movef(dtko, dtk, timo);

            // wall temperature / velocity condition

            // wall_condition_update(d_qa, d_npex, umf, dcomp_info, nbc);
        }

        // advance in time
        n++;
        timo += dt;

        // record intermediate results
        // if(output_enabled)
        // {
        //     if(timo > (-tmax) / ndata)
        //     {
        //         dtsum += dt;
        //         if(nout == 1)
        //         {

        //         }
        //     }
        // }

    } while(timo < tmax && (dt != 0.0f || n <= 2));

    for(int i=0; i<5; i++) hipStreamDestroy(stream[i]);

    // Finalize the MPI environment.
    check_mpi(MPI_Finalize());

    return 0;
}
